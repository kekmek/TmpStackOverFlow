#include <iostream>
#include <random>
#include <chrono>

#include <hip/hip_runtime.h>

const int num_seeds = 20;

__global__ void vectorAdd(float* a, float* b, float* c, int vector_size) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < vector_size) {
        c[index] = a[index] + b[index];
    }
}

void printMatrix(float* a, int n) {
    for (int i = 0; i < n; ++i) {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;
}

int main(int argc, char** argv) {

    int N = std::stoi(argv[1]);

    for (int j = 0; j < num_seeds; ++j) {

        int size = N * sizeof(float);

        float* host_a = nullptr;
        float* host_b = nullptr;
        float* host_c = nullptr;

        // cudaMallocHost((void**)& host_a, size);
        // cudaMallocHost((void**)& host_b, size);   
        // cudaMallocHost((void**)& host_c, size);

        host_a = new float[N]{};
        host_b = new float[N]{};
        host_c = new float[N]{};

        float* device_a, *device_b, *device_c;

        std::random_device rd;
        std::mt19937 mt(rd());
        std::uniform_real_distribution<float> dist;

        for (int i = 0; i < N; ++i) {
            host_a[i] = dist(mt);
            host_b[i] = dist(mt);
        }

        dim3 blockDim(1024, 1, 1);
        dim3 dimGrid((N + blockDim.x - 1) / blockDim.x, 1, 1);

        hipMalloc((void**)&device_a, size);
        hipMalloc((void**)&device_b, size);
        hipMalloc((void**)&device_c, size);

        hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
        hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);

        vectorAdd<<<dimGrid, blockDim>>>(device_a, device_b, device_c, N);
        
        hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);
        
        // printMatrix(host_a, N);
        // printMatrix(host_b, N);
        // printMatrix(host_c, N);

        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);

        // cudaFreeHost(host_a);
        // cudaFreeHost(host_b);
        // cudaFreeHost(host_c);

        delete[] host_a;
        delete[] host_b;
        delete[] host_c;
    }
    return 0;
}